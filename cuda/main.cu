#include "hip/hip_runtime.h"
#include <ctime>
#include "type.h"
#include "auxiliary.h"
#include "screen.h"
#include "scene_loader.h"
#include "raytracer.h"

namespace render {

#ifdef ENABLE_GPU

static ENTRY void init_(hiprandState *rand_state, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = y * width + x;
	if (idx >= width * height) return ;
	hiprand_init(1984, idx, 0, &rand_state[idx]);
}

static ENTRY void render(hiprandState *rand_state, raytracer *r, scene *sc, screen *scrn)
{
	thread_ctx ctx;
	raytracer R = *r;
	float width = scrn->getsize().x();
	float height = scrn->getsize().y();
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return; 
	int idx = y * width + x;
	if (idx >= width * height) return ;
	ctx.rnd = rand_state[idx];
	ctx.x = x;
	ctx.y = y;
	R.render(&ctx, *sc, *scrn);
}

#else

static random_ctx_t *init_(int width, int height) 
{

}

#endif

int main(int argc, const char *argv[])
{
#if 1
	int key = 0;
	if (argc < 2) {
		fprintf(stderr, "USAGE:%s scene\n", argv[0]);
		exit(0);
	}

	raytracer *R = nullptr;

	scene_loader sl;
	sl.load(argv[1]);

	auto &conf_screen = sl.getscreen();
	screen *scrn = new screen(conf_screen.width, conf_screen.height);

	std::cout << "scene:" << sizeof(scene) << std::endl;
	scene *scene1 = new scene();
	scene1->add(sl.getprimitives());

	auto &conf_cam= sl.getcamera();
	camera *cam = new camera(conf_cam.eye, conf_cam.fov, conf_cam.znear, conf_cam.zfar);
	cam->yaw(conf_cam.yaw);

	auto &bg = sl.getbackground();

	switch (sl.getrender()) {
	case scene_loader::RASTERIZE:
		assert(0);
		break;
	case scene_loader::RAYTRACING:
		R = new raytracer(*cam, raytracer::RAYTRACING, bg);
		break;
	case scene_loader::PATHTRACING:
		R = new raytracer(*cam, raytracer::PATHTRACING, bg);
		break;
	default:
		assert(0);
	}

	int nx = scrn->getsize().x();
	int ny = scrn->getsize().y();
	int tx = 8;
	int ty = 8;

	int width = nx;
	int height = ny;

	dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
	dim3 threads(tx, ty);
	
	hiprandState *rand_state;
	checkX(hipMalloc(&rand_state, sizeof(hiprandState) * width * height));

	GPU_CALL(blocks, threads, init_, rand_state, width, height);

	if (argc == 2) {
		int spp = 1;
		while(key != 27)
		{
			scrn->clear();
			scrn->scale(1.f / spp);
			clock_t start, stop;
			start = clock();
			GPU_CALL(blocks, threads, render, rand_state, R, scene1, scrn);
			stop = clock();
			std::cout << "takes:" << (double)(stop - start) / CLOCKS_PER_SEC << "s" << std::endl;
			scrn->show();
			key = cv::waitKey(1);
			std::cout << "key:" << key << std::endl << std::endl;
			switch (key) {
			case 'w':
				spp = 1;
				scrn->clear();
				std::cout << "w" << std::endl << std::endl;
				cam->move(10.f);
				break;
			case 's':
				spp = 1;
				scrn->clear();
				std::cout << "s" << std::endl << std::endl;
				cam->move(-10.f);
				break;
			case 'a':
				spp = 1;
				scrn->clear();
				std::cout << "a" << std::endl << std::endl;
				cam->yaw(1.f);
				break;
			case 'd':
				spp = 1;
				scrn->clear();
				std::cout << "d" << std::endl << std::endl;
				cam->yaw(-1.f);
				break;
			}
		}
	} else {
		/*
		int spp = sl.getspp();
			while (!R->render(scene1, scrn, spp));
			scrn.dump("out.ppm");
		*/
	}

	delete R;
	delete scrn;
	delete scene1;
	delete cam;
#endif
	return 0;
}

}


int main(int argc, const char *argv[]) 
{
	return render::main(argc, argv);
}

